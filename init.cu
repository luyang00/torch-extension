#include <luaT.h>
#include <lua.h>

#include <THC.h>
#include <THCGeneral.h>

THCState* getTorchState(lua_State* state) {
	THCState* torch = NULL;

	lua_getglobal(state, "cutorch");
	lua_getfield(state, -1, "getState");
	lua_call(state, 0, 1);
	torch = (THCState*) lua_touserdata(state, -1);
	lua_pop(state, 2);

	return torch;
}

#include "src/HadamardProduct_cuda.c"

LUA_EXTERNC DLL_EXPORT int luaopen_libnnex(lua_State* state);

int luaopen_libnnex(lua_State* state) {
	lua_newtable(state);

	HadamardProduct_cuda_init(state);

	return 1;
}